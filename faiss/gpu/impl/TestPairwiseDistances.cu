
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/PairwiseDistances.cuh>
#include <faiss/gpu/impl/SelfPairwiseDistances.cuh>
#include <faiss/gpu/impl/TestPairwiseDistances.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/MathOperators.cuh>
#include <faiss/gpu/utils/PtxUtils.cuh>
#include <faiss/gpu/utils/Reductions.cuh>
#include <algorithm>


namespace faiss {
  namespace gpu {

    namespace test_pwd { // namespace for test cases
      
      //
      // Test cases
      //

      template<typename T>
      void test_case_0(Tensor<T, 5, true, int>& dists,
		       Tensor<T, 4, true, int>& burst,
		       Tensor<int, 5, true, int>& blocks,
		       Tensor<T, 5, true, int>& centroids,
		       int patchsize, float offset,
		       hipStream_t stream){
	T* one = (T*)malloc(sizeof(T));
	*one = 1;
	for (int i0 = 0; i0 < dists.getSize(0); ++i0){
	  for (int i1 = 0; i1 < dists.getSize(1); ++i1){
	    for (int i2 = 0; i2 < dists.getSize(2); ++i2){
	      for (int i3 = 0; i3 < dists.getSize(3); ++i3){
		for (int i4 = 0; i4 < dists.getSize(4); ++i4){
		  hipMemcpy(dists[i0][i1][i2][i3][i4].data(),one,
			     sizeof(T),hipMemcpyHostToDevice);
		}
	      }
	    }
	  }
	}
	free(one);
      }

      template<typename T>
      void test_case_1(Tensor<T, 5, true, int>& dists,
		       Tensor<T, 4, true, int>& burst,
		       Tensor<int, 5, true, int>& blocks,
		       Tensor<T, 5, true, int>& centroids,
		       int patchsize, float offset,
		       hipStream_t stream){
	self_pairwise_distances(dists,burst,blocks,patchsize,offset,stream);
      }

      template<typename T>
      void test_case_2(Tensor<T, 5, true, int>& dists,
		       Tensor<T, 4, true, int>& burst,
		       Tensor<int, 5, true, int>& blocks,
		       Tensor<T, 5, true, int>& centroids,
		       int patchsize, float offset,
		       hipStream_t stream){
	pairwise_distances(dists,burst,blocks,centroids,
			   patchsize,offset,stream);
      }

    } // namespace test_pwd
    

    //
    // Main Test Function 
    //

    template<typename T>
    void test_pairwise_distances(int test_case,
				 Tensor<T, 5, true, int>& dists,
				 Tensor<T, 5, true, int>& self_dists,
				 Tensor<T, 4, true, int>& burst,
				 Tensor<int, 5, true, int>& blocks,
				 Tensor<T, 5, true, int>& centroids,
				 int patchsize, float offset,
				 hipStream_t stream){

      fprintf(stdout,"Testing: [pairwise dists.]\n");
      if (test_case == 0){
	test_pwd::test_case_0<T>(dists,burst,blocks,centroids,
				 patchsize,offset,stream);
      }else if (test_case == 1){
	test_pwd::test_case_1<T>(self_dists,burst,blocks,centroids,
				 patchsize,offset,stream);
      }else if (test_case == 2){
	test_pwd::test_case_2<T>(dists,burst,blocks,centroids,
				 patchsize,offset,stream);
      }else{
	FAISS_THROW_FMT("[TestPairwiseDistances.cu]: unimplemented test case %d",
			test_case);
      }
    }


    //
    // Template Inits
    //
    
    void test_pairwise_distances(int test_case,
				 Tensor<float, 5, true, int>& dists,
				 Tensor<float, 5, true, int>& self_dists,
				 Tensor<float, 4, true, int>& burst,
				 Tensor<int, 5, true, int>& blocks,
				 Tensor<float, 5, true, int>& centroids,
				 int patchsize, float offset,
				 hipStream_t stream){
      test_pairwise_distances<float>(test_case,dists,self_dists,
				     burst,blocks,centroids,
				     patchsize, offset, stream);
    }

    void test_pairwise_distances(int test_case,
				 Tensor<half, 5, true, int>& dists,
				 Tensor<half, 5, true, int>& self_dists,
				 Tensor<half, 4, true, int>& burst,
				 Tensor<int, 5, true, int>& blocks,
				 Tensor<half, 5, true, int>& centroids,
				 int patchsize, float offset,
				 hipStream_t stream){
      test_pairwise_distances<half>(test_case,dists,self_dists,
				    burst,blocks,centroids,
				    patchsize, offset, stream);
    }

  }
}
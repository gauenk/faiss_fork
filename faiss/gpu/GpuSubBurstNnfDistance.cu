#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuSubBurstNnfDistance.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/SubBurstPatchDistance.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>

namespace faiss {
  namespace gpu {

    template <typename T>
    void bfSubBurstNnfConvert(GpuResourcesProvider* prov,
			   const GpuSubBurstNnfDistanceParams& args) {
      // Validate the input data
      // FAISS_THROW_IF_NOT_MSG(
      //         args.k > 0 || args.k == -1,
      //         "bfSubBurstNnf: k must be > 0 for top-k reduction, "
      //         "or -1 for all pairwise distances");
      // FAISS_THROW_IF_NOT_MSG(args.dims > 0, "bfSubBurstNnf: dims must be > 0");
      // FAISS_THROW_IF_NOT_MSG(
      //         args.numVectors > 0, "bfSubBurstNnf: numVectors must be > 0");
      // FAISS_THROW_IF_NOT_MSG(
      //         args.vectors, "bfSubBurstNnf: vectors must be provided (passed null)");
      // FAISS_THROW_IF_NOT_MSG(
      //         args.numQueries > 0, "bfSubBurstNnf: numQueries must be > 0");
      // FAISS_THROW_IF_NOT_MSG(
      //         args.queries, "bfSubBurstNnf: queries must be provided (passed null)");
      FAISS_THROW_IF_NOT_MSG(
              args.outDistances,
              "bfSubBurstNnf: outDistances must be provided (passed null)");
      FAISS_THROW_IF_NOT_MSG(
              args.outIndices || args.k == -1,
              "bfSubBurstNnf: outIndices must be provided (passed null)");

      // Don't let the resources go out of scope
      // std::cout << "about to get res" << std::endl;
      auto resImpl = prov->getResources();
      auto res = resImpl.get();
      // std::cout << "res" << std::endl;
      auto device = getCurrentDevice();
      auto stream = res->getDefaultStreamCurrentDevice();
      // std::cout << "Got the Stream!" << std::endl;

      int psHalf = std::floor(args.ps/2);
      int pad = std::floor(args.ps/2) + std::floor(args.nblocks/2);
      auto burst = toDeviceTemporary<T, 4>(res,device,
					   const_cast<T*>(reinterpret_cast<const T*>
							  (args.burst)),
					   stream,
					   {args.sub_t,args.c,args.h+2*pad,args.w+2*pad});
      auto subAve = toDeviceTemporary<T, 3>(res,device,
						const_cast<T*>(
						reinterpret_cast<const T*>
							       (args.subAve)),
						stream,
						{args.c,args.h+2*psHalf,args.w+2*psHalf});
      auto blockLabels = toDeviceTemporary<int, 5>(res,device,
					    args.blockLabels,
					    stream,
					    {args.nblocks_total,
					       args.h+2*psHalf,args.w+2*psHalf,
					       args.sub_t,2});
      auto mask = toDeviceTemporary<bool, 4>(res,device,
					     const_cast<bool*>(
                                             reinterpret_cast<const bool*>
							    (args.mask)),
					    stream,
					    {args.nblocks_total,
					       args.h+2*psHalf,args.w+2*psHalf,
					       args.sub_t});
      auto tOutDistances = toDeviceTemporary<float, 3>(
						       res,
						       device,
						       args.outDistances,
						       stream,
						       {args.h,args.w,args.k});

      if (args.outIndicesType == IndicesDataType::I64) {
        // The brute-force API only supports an interface for i32 indices only,
        // so we must create an output i32 buffer then convert back
        DeviceTensor<int, 5, true> tOutIntIndices(res,
						  makeTempAlloc(AllocType::Other, stream),
						  {args.sub_t, args.h, args.w, args.k, 2});

        // Since we've guaranteed that all arguments are on device, call the
        // implementation

        bfSubBurstNnfOnDevice<T>(
                res,
		device,
                stream,
		burst,
		subAve,
		blockLabels,
		mask,
                args.k,
		args.t,
		args.h,
		args.w,
		args.c,
		args.ps,
		args.nblocks,
		args.valMean,
                args.metric,
                args.metricArg,
                tOutDistances,
                tOutIntIndices,
                args.ignoreOutDistances);

        // Convert and copy int indices out
        auto tOutIndices = toDeviceTemporary<Index::idx_t, 5>(res,device,
							      (Index::idx_t*)
							      args.outIndices,
							      stream,
							      {args.sub_t, args.h, args.w,
								 args.k, 2});

        // Convert int to idx_t
        convertTensor<int, Index::idx_t, 5>(stream, tOutIntIndices, tOutIndices);

        // Copy back if necessary
        fromDevice<Index::idx_t, 5>(tOutIndices, (Index::idx_t*)args.outIndices, stream);

      } else if (args.outIndicesType == IndicesDataType::I32) {
        // We can use the brute-force API directly, as it takes i32 indices
        // FIXME: convert to int32_t everywhere?
        static_assert(sizeof(int) == 4, "");

        auto tOutIntIndices = toDeviceTemporary<int, 5>(res,device,
							(int*)args.outIndices,
							stream,
							{args.sub_t, args.h,
							 args.w, args.k, 2});

        // Since we've guaranteed that all arguments are on device, call the
        // implementation
        bfSubBurstNnfOnDevice<T>(
                res,
		device,
                stream,
		burst,
		subAve,
		blockLabels,
		mask,
                args.k,
		args.t,
                args.h,
                args.w,
                args.c,
		args.ps,
		args.nblocks,
		args.valMean,
                args.metric,
                args.metricArg,
                tOutDistances,
                tOutIntIndices,
                args.ignoreOutDistances);

        // Copy back if necessary
        fromDevice<int, 5>(tOutIntIndices, (int*)args.outIndices, stream);
      } else {
        FAISS_THROW_MSG("unknown outIndicesType");
      }

      // Copy distances back if necessary
      fromDevice<float, 3>(tOutDistances, args.outDistances, stream);
    }

    void bfSubBurstNnf(GpuResourcesProvider* res,
		       const GpuSubBurstNnfDistanceParams& args) {
      // For now, both vectors and queries must be of the same data type

      if (args.dType == DistanceDataType::F32) {
	bfSubBurstNnfConvert<float>(res, args);
      } else if (args.dType == DistanceDataType::F16) {
      	bfSubBurstNnfConvert<half>(res, args);
      } else {
        FAISS_THROW_MSG("unknown vectorType");
      }
    }


  }
}
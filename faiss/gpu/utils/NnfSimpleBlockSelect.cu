#include "hip/hip_runtime.h"
/**
 * Copyright (c) Kent Gauen
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#pragma once

#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/gpu/utils/MathOperators.cuh>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/utils/NnfSimpleBlockSelect.cuh>

/****
     Select "topK" from "blockTileSize" of inVals
 ****/

#define ABS(N) ((N<0)?(-N):(N))

namespace faiss {
  namespace gpu {

    __global__ void nnfBlockSelect(
				   Tensor<float, 3, true> inVals,
				   Tensor<int, 2, true> inKeys,
				   Tensor<float, 3, true> outVals,
				   Tensor<int, 4, true> outKeys,
				   float valMean) {
      int row = threadIdx.x + blockDim.x * blockIdx.x;
      int col = threadIdx.y + blockDim.y * blockIdx.y;
      int numOfComps = inVals.getSize(2);
      bool legal_row = row < inVals.getSize(0);
      bool legal_col = col < inVals.getSize(1);
      int k = outVals.getSize(2);
      int kidx = 0;

      if ( legal_row && legal_col ) {

	float outVal_max = outVals[row][col][k-1]; // already corrected value
	float outVal_curr = outVal_max;
	for (int comp = 0; comp < numOfComps; ++comp){

	  float inVal = ABS(inVals[row][col][comp] - valMean);

	  if (inVal < outVal_max){
	    kidx = k-1;
	    outVal_curr = outVal_max;
	    while( inVal < outVal_curr && kidx > 0){
	      kidx -= 1;
	      outVal_curr = outVals[row][col][kidx];
	    }
	    if (kidx != 0){ kidx += 1; }
	    else if (inVal > outVal_curr){ kidx += 1; }
	    // printf("an assign!: %d,%f\n",kidx,inVal);

	    // shift values up
	    for (int sidx = k-1; sidx > kidx; --sidx){
	      outVals[row][col][sidx] = (float)outVals[row][col][sidx-1];
	      outKeys[row][col][sidx][0] = (int)outKeys[row][col][sidx-1][0];
	      outKeys[row][col][sidx][1] = (int)outKeys[row][col][sidx-1][1];
	    }

	    // assign new values
	    outVals[row][col][kidx] = inVal;
	    outKeys[row][col][kidx][0] = inKeys[comp][0];
	    outKeys[row][col][kidx][1] = inKeys[comp][1];
	    outVal_max = outVals[row][col][k-1];

	  }
	}
      }
    }
    
    void runNnfSimpleBlockSelect(
        Tensor<float, 3, true>& inVals,
        Tensor<int, 2, true>& inKeys,
        Tensor<float, 3, true>& outVals,
        Tensor<int, 4, true>& outKeys,
	float valMean, bool comp_with_out,int k,
        hipStream_t stream){

      // assert shapes 
      FAISS_ASSERT(inVals.getSize(2) == inKeys.getSize(0));
      FAISS_ASSERT(outVals.getSize(0) == outKeys.getSize(0));
      FAISS_ASSERT(outVals.getSize(1) == outKeys.getSize(1));
      FAISS_ASSERT(inVals.getSize(0) == outVals.getSize(0));
      FAISS_ASSERT(inVals.getSize(1) == outVals.getSize(1));
      FAISS_ASSERT(outKeys.getSize(0) == outVals.getSize(0));
      FAISS_ASSERT(outKeys.getSize(1) == outVals.getSize(1));
      FAISS_ASSERT(outVals.getSize(2) == k);
      FAISS_ASSERT(outKeys.getSize(2) == k);
      
      // setup kernel launch
      // keep it simple; each (h,w) index gets a thread, _not_ a block
      // it is not as parallel as it could be. 
      // this will probably have horrible warp divergence too
      int maxThreads = (int) getMaxThreadsCurrentDevice();
      // std::cout << "maxThreads: " << maxThreads << std::endl;
      int sqrtThreads = 32;//utils::pow(maxThreads*1.0, .5);
      
      auto nBlocksH = utils::divUp(inVals.getSize(0),sqrtThreads);
      auto nBlocksW = utils::divUp(inVals.getSize(1),sqrtThreads);
      
      // printf("(nBlocksH,nBlocksW,sqrtThreads): (%d,%d,%d)\n",nBlocksH,nBlocksW,sqrtThreads);
      auto grid = dim3(nBlocksH,nBlocksW);
      auto block = dim3(sqrtThreads,sqrtThreads);

      // launch kernel
      nnfBlockSelect<<<grid, block, 0, stream>>>(inVals, inKeys,
						 outVals, outKeys,
						 valMean);
      CUDA_TEST_ERROR();
    }
    
  }
}
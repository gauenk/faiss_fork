/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/burstblockselect/BurstBlockSelectImpl.cuh>

namespace faiss {
namespace gpu {

// warp Q to thread Q:
// 1, 1
// 32, 2
// 64, 3
// 128, 3
// 256, 4
// 512, 8
// 1024, 8
// 2048, 8

BURST_BLOCK_SELECT_DECL(float, true, 1);
BURST_BLOCK_SELECT_DECL(float, true, 32);
BURST_BLOCK_SELECT_DECL(float, true, 64);
BURST_BLOCK_SELECT_DECL(float, true, 128);
BURST_BLOCK_SELECT_DECL(float, true, 256);
BURST_BLOCK_SELECT_DECL(float, true, 512);
BURST_BLOCK_SELECT_DECL(float, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
BURST_BLOCK_SELECT_DECL(float, true, 2048);
#endif

BURST_BLOCK_SELECT_DECL(float, false, 1);
BURST_BLOCK_SELECT_DECL(float, false, 32);
BURST_BLOCK_SELECT_DECL(float, false, 64);
BURST_BLOCK_SELECT_DECL(float, false, 128);
BURST_BLOCK_SELECT_DECL(float, false, 256);
BURST_BLOCK_SELECT_DECL(float, false, 512);
BURST_BLOCK_SELECT_DECL(float, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
BURST_BLOCK_SELECT_DECL(float, false, 2048);
#endif

void runBurstBlockSelect(
        Tensor<float, 3, true>& in,
        Tensor<float, 3, true>& outK,
        Tensor<int, 3, true>& outV,
        bool dir,
        int k,
        hipStream_t stream) {
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BURST_BLOCK_SELECT_CALL(float, true, 1);
        } else if (k <= 32) {
            BURST_BLOCK_SELECT_CALL(float, true, 32);
        } else if (k <= 64) {
            BURST_BLOCK_SELECT_CALL(float, true, 64);
        } else if (k <= 128) {
            BURST_BLOCK_SELECT_CALL(float, true, 128);
        } else if (k <= 256) {
            BURST_BLOCK_SELECT_CALL(float, true, 256);
        } else if (k <= 512) {
            BURST_BLOCK_SELECT_CALL(float, true, 512);
        } else if (k <= 1024) {
            BURST_BLOCK_SELECT_CALL(float, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BURST_BLOCK_SELECT_CALL(float, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BURST_BLOCK_SELECT_CALL(float, false, 1);
        } else if (k <= 32) {
            BURST_BLOCK_SELECT_CALL(float, false, 32);
        } else if (k <= 64) {
            BURST_BLOCK_SELECT_CALL(float, false, 64);
        } else if (k <= 128) {
            BURST_BLOCK_SELECT_CALL(float, false, 128);
        } else if (k <= 256) {
            BURST_BLOCK_SELECT_CALL(float, false, 256);
        } else if (k <= 512) {
            BURST_BLOCK_SELECT_CALL(float, false, 512);
        } else if (k <= 1024) {
            BURST_BLOCK_SELECT_CALL(float, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BURST_BLOCK_SELECT_CALL(float, false, 2048);
#endif
        }
    }
}

void runBurstBlockSelectPair(
        Tensor<float, 3, true>& inK,
        Tensor<int, 3, true>& inV,
        Tensor<float, 3, true>& outK,
        Tensor<int, 3, true>& outV,
        bool dir,
        int k,
        hipStream_t stream) {
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 1);
        } else if (k <= 32) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 32);
        } else if (k <= 64) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 64);
        } else if (k <= 128) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 128);
        } else if (k <= 256) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 256);
        } else if (k <= 512) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 512);
        } else if (k <= 1024) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 1);
        } else if (k <= 32) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 32);
        } else if (k <= 64) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 64);
        } else if (k <= 128) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 128);
        } else if (k <= 256) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 256);
        } else if (k <= 512) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 512);
        } else if (k <= 1024) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BURST_BLOCK_SELECT_PAIR_CALL(float, false, 2048);
#endif
        }
    }
}

} // namespace gpu
} // namespace faiss
